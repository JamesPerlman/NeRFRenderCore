#include "hip/hip_runtime.h"

#include "gpu-image.cuh"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stbi/stb_image.h>
#include <stbi/stb_image_write.h>
#include <vector>
#include <memory>

#include "parallel-utils.cuh"

NRC_NAMESPACE_BEGIN

__global__ void buffer_to_stbi_uc(
    const uint32_t n_elements,
    const float* __restrict__ input,
    stbi_uc* __restrict__ output,
    const float scale
) {
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < n_elements) {
        output[idx] = (stbi_uc)((float)input[idx] / scale * 255.0f);
    }
}

template <typename T>
__global__ void join_channels_kernel(
    const uint32_t n_pixels,
    const int n_channels,
    const T* __restrict__ input,
    T* __restrict__ output
) {
    const uint32_t pix_idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (pix_idx >= n_pixels) return;

    const uint32_t j_idx = n_channels * pix_idx;

    uint32_t c_idx = pix_idx;

    for (int i = 0; i < n_channels; ++i) {
        output[j_idx + i] = input[c_idx];
        c_idx += n_pixels;
    }
}

void save_buffer_to_image(
    const hipStream_t& stream,
    const std::string& filename,
    const float* data,
    const uint32_t& width,
    const uint32_t& height,
    const uint32_t channels,
    std::thread& imgWriteThread,
    const uint32_t& stride,
    const float& scale
) {
    const uint32_t n_pixels = width * height;
    const uint32_t n_elements = n_pixels * channels;

    tcnn::GPUMemory<stbi_uc> img_gpu(n_elements);
    tcnn::GPUMemory<float> data_float(n_elements);

    auto img_cpu = std::make_unique<std::vector<stbi_uc>>(n_elements);

    if (std::is_same<float, stbi_uc>::value) {
        CUDA_CHECK_THROW(hipMemcpyAsync((*img_cpu).data(), data, n_elements * sizeof(stbi_uc), hipMemcpyDeviceToHost, stream));
    } else {

        copy_and_cast(stream, n_elements, data_float.data(), data);

        buffer_to_stbi_uc<<<tcnn::n_blocks_linear(n_elements), tcnn::n_threads_linear, 0, stream>>>(n_elements, data_float.data(), img_gpu.data(), scale);

        if (stride > 0) {
            tcnn::GPUMemory<stbi_uc> img_gpu_decontig(n_elements);
            join_channels_kernel<<<tcnn::n_blocks_linear(n_pixels), tcnn::n_threads_linear, 0, stream>>>(n_pixels, channels, img_gpu.data(), img_gpu_decontig.data());
            img_gpu_decontig.copy_to_host(*img_cpu);
        } else {
            img_gpu.copy_to_host(*img_cpu);
        }
    }

    auto f = [](std::string filename, uint32_t width, uint32_t height, uint32_t channels, std::unique_ptr<std::vector<stbi_uc>> imgcpu) {
        stbi_write_png(filename.c_str(), width, height, channels, (*imgcpu).data(), width * sizeof(stbi_uc) * channels);
    };
    imgWriteThread = std::thread(f, filename, width, height, channels, std::move(img_cpu));
}

std::vector<float> save_buffer_to_memory(
    const hipStream_t& stream,
    const float* data,
    const uint32_t& width,
    const uint32_t& height,
    const uint32_t& channels,
    const uint32_t& stride,
    const float& scale
) {
    const uint32_t n_pixels = width * height;
    const uint32_t n_elements = n_pixels * channels;


    std::vector<float> img_cpu(n_elements);

    if (stride > 0) {
        tcnn::GPUMemory<float> img_gpu_decontig(n_elements);
        join_channels_kernel<<<tcnn::n_blocks_linear(n_pixels), tcnn::n_threads_linear, 0, stream>>>(n_pixels, channels, data, img_gpu_decontig.data());
        img_gpu_decontig.copy_to_host(img_cpu);
    } else {
        hipMemcpy(img_cpu.data(), data, n_elements * sizeof(float), hipMemcpyDeviceToHost);
    }

    return img_cpu;
}

void join_channels(
    const hipStream_t& stream,
    const uint32_t& width,
    const uint32_t& height,
    const uint32_t& channels,
    const float* data,
    float* output
) {
    const uint32_t n_pixels = width * height;

    join_channels_kernel<<<tcnn::n_blocks_linear(n_pixels), tcnn::n_threads_linear, 0, stream>>>(n_pixels, channels, data, output);
}

NRC_NAMESPACE_END
